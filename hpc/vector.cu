#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdbool.h> // Include for bool type

__global__ void vectorAdd(int* a, int* b, int* c, int size) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

void vectorAddNormal(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

// Function to compare results of CUDA and normal vector addition
bool verifyResults(int* c_cuda, int* c_normal, int size) {
    for (int i = 0; i < size; i++) {
        if (c_cuda[i] != c_normal[i]) {
            printf("Mismatch at index %d: CUDA result = %d, Normal result = %d\n", i, c_cuda[i], c_normal[i]);
            return false;
        }
    }
    return true;
}

int main() 
{
    int size = 1000000; 
    int* a, * b, * c_cuda, * c_normal;    
    int* dev_a, * dev_b, * dev_c;  

    // Allocate memory for host vectors
    a = (int*)malloc(size * sizeof(int));
    b = (int*)malloc(size * sizeof(int));
    c_cuda = (int*)malloc(size * sizeof(int));
    c_normal = (int*)malloc(size * sizeof(int));

    // Initialize host vectors
    for (int i = 0; i < size; i++) {
        a[i] = rand() % 1000;
        b[i] = rand() % 1000;
    }

    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    clock_t start_cuda, end_cuda;
    start_cuda = clock();

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    vectorAdd<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);

    hipDeviceSynchronize();
    end_cuda = clock();
    double cuda_time = (double)(end_cuda - start_cuda) / CLOCKS_PER_SEC;

    hipMemcpy(c_cuda, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("CUDA time: %f", cuda_time);  

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    clock_t start_normal, end_normal;
    start_normal = clock();

    vectorAddNormal(a, b, c_normal, size);

    end_normal = clock();
    double normal_time = (double)(end_normal - start_normal) / CLOCKS_PER_SEC;

    printf("\nNormal time: %f ", normal_time);  

    // Compare outputs
    bool is_correct = verifyResults(c_cuda, c_normal, size);
    printf("\nOutput Match: %s", is_correct ? "True" : "False");

    // Calculate speedup
    double speedup = normal_time / cuda_time;
    printf("\nSpeedup (Normal / CUDA): %f\n", speedup);

    free(a);
    free(b);
    free(c_cuda);
    free(c_normal);

    return 0;
}
